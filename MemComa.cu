/*
1. Memory Copy Cost   One-Step
2. Straggler: Ring-based
**/
#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <vector>
#include <map>
#include <fstream>
#include <algorithm>
using namespace std;

#define Malloc(type,n) (type *)malloc((n)*sizeof(type))

static void HandleError( hipError_t err, const char *file, int line )
{
	if (err != hipSuccess)
	{
		printf( "%s in %s at line %d\n", hipGetErrorString( err ), file, line );
		exit(-1);
	}
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))
struct Block
{
	int block_id;
	int data_age;
	int sta_idx;
	int height; //height
	int ele_num;
	bool isP;
	double* eles;
	Block()
	{

	}
	Block operator=(Block& bitem)
	{
		block_id = bitem.block_id;
		data_age = bitem.data_age;
		height = bitem.height;
		eles = bitem.eles;
		ele_num = bitem.ele_num;
		sta_idx = bitem.sta_idx;
		return *this;
	}
	void printBlock()
	{

		printf("block_id  %d\n", block_id);
		printf("data_age  %d\n", data_age);
		printf("ele_num  %d\n", ele_num);
		for (int i = 0; i < ele_num; i++)
		{
			printf("%lf\t", eles[i]);
		}
		printf("\n");

	}
};
struct RatingEntry{
	int pidx;
	int qidx;
	double rate;
	RatingEntry(){

	}
	RatingEntry(int p, int q, double r){
		pidx = p;
		qidx = q;
		rate = r;
	}
	RatingEntry operator=(RatingEntry& ritem){
		pidx = ritem.pidx;
		qidx = ritem.qidx;
		rate = ritem.rate;
		return *this;
	}
};

//Yahoo!Music
#define FILE_NAME "./trainDS/"
#define TEST_NAME "./testDS"
#define N 1000990
#define M 624961
#define K 100 //主题个数
double yita = 0.001;
double theta = 0.05;


//#define SM_NUM 8
//#define MM_NUM 4

#define SM_NUM 4
#define MM_NUM 1
#define BK_NUM (SM_NUM*MM_NUM)
#define TD_NUM 1
#define BT_NUM (BK_NUM*TD_NUM)
#define RB_NUM (BT_NUM*BT_NUM)
#define ITER_CAP 500
Block PBlocks[BK_NUM], QBlocks[BK_NUM];
int random_seq[TD_NUM * ITER_CAP];
Block *dev_PBlocks, *dev_QBlocks;
int* dev_seq;
int* dev_flag;
double* dev_PData[BK_NUM], *dev_QData[BK_NUM];
double *dev_p_cache[BT_NUM], *dev_q_cache[BT_NUM];
std::map<long, double> EntryM;
vector<RatingEntry> Rblocks[BK_NUM*TD_NUM][BK_NUM*TD_NUM];
RatingEntry* dev_rate_entries[RB_NUM];
double entry_num[RB_NUM];
int* dev_entry_num;
int p_height, q_height, t_p_height, t_q_height;

void readTrainData();
void initParas();
void allocCudaMem();
void partitionP(int portion_num, int line_num,  Block * block_arr);
void freeCudaMem();

__global__ void MFkernel(Block* dev_PBlocks, Block* dev_QBlocks, double* dev_PData[], double* dev_QData[], double *dev_p_cache[],double *dev_q_cache[], int* dev_seq, int*dev_entry_num, RatingEntry* dev_rate_entries[], int p_height, int q_height, int* dev_flag, int epoch, double yita, double theta)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int tid = threadIdx.x;
	int bid = blockIdx.x;
	int iden = bid*TD_NUM+tid;
	printf("Test %d  %d  %d %d\n", tid, bid, i, iden );
	int iter = 0;

	Block* pblock = &(dev_PBlocks[bid]);
	Block* qblock = &(dev_QBlocks[BK_NUM-bid-1]);
	pblock->eles = dev_PData[bid];
	qblock->eles = dev_QData[BK_NUM-bid-1];
	//int row_unit = (pblock->ele_num)/TD_NUM;
	//int col_unit = (qblock->ele_num)/TD_NUM;
	int row, col, row_b_id, col_b_id, row_base, col_base, mini_b_idx, td_ele_num;
	int ele_idx, ele_p, ele_q, dimk;
	double ele_ra, error, sum_ra;
	RatingEntry* td_rate_entries;
	
	// one epoch
	row_b_id = bid;
	col_b_id = (BK_NUM - 1-row_b_id+epoch)%BK_NUM;

	row_base = row_b_id * BT_NUM;
	col_base = col_b_id *BT_NUM;

	for (iter = 0; iter < ITER_CAP; iter++)
	{
		//SGD
	 	row = row_base + tid ;
		col = col_base + dev_seq[iter*TD_NUM+tid];
		mini_b_idx = row*BT_NUM+col;
		td_rate_entries = dev_rate_entries[mini_b_idx];
		td_ele_num = dev_entry_num[mini_b_idx];

		for(ele_idx = 0; ele_idx<td_ele_num; ele_idx++){
			ele_p = (td_rate_entries[ele_idx].pidx)%p_height;
			ele_q = (td_rate_entries[ele_idx].qidx)%q_height;
			ele_ra = td_rate_entries[ele_idx].rate;
			sum_ra = 0;
			for(dimk = 0; dimk<K; dimk++){
				sum_ra+= dev_PData[row_b_id][ele_p*K+dimk]*dev_QData[col_b_id][ele_q*K+dimk];
				dev_p_cache[iden][dimk] = dev_PData[row_b_id][ele_p*K+dimk];
				dev_q_cache[iden][dimk] = dev_QData[col_b_id][ele_q*K+dimk];
			}
			error = ele_ra -sum_ra;
			//SGD
			for(dimk = 0; dimk < K; dimk++){
				dev_PData[row_b_id][ele_p*K+dimk] += yita * (error * dev_q_cache[iden][dimk] - theta * dev_p_cache[iden][dimk]);
				dev_QData[col_b_id][ele_q*K+dimk] += yita * (error * dev_p_cache[iden][dimk] - theta * dev_q_cache[iden][dimk]);
			}
		}
		//Sync
		__syncthreads();

	}

}

__global__ void helloFromGPU(void)
{
	printf("Hello from GPU\n");
}

void readTrainData()
{
	char fn[100];
	int i, j;
	for ( i = 0; i < 64; i++)
	{
		for (j = 0; j < 64; j++)
		{
			//iidx = i / (64 / BK_NUM);
			//jidx = j / (64 / BK_NUM);
			sprintf(fn, "%s%d-%d", FILE_NAME, i, j);
			ifstream ifs(fn);
			if (!ifs.is_open())
			{
				printf("Open fail %s\n", fn);
				exit(-1);
			}
			long hash_idx = -1;
			double ra = 0;
			while (!ifs.eof())
			{
				ifs >> hash_idx >> ra;
				if (hash_idx >= 0)
				{
					ra = ra / 100.0;
					//Rblocks[iidx][jidx].insert(pair<long, double>(hash_idx, ra));
					EntryM.insert(pair<long, double>(hash_idx, ra));

				}
			}

		}
		printf("row %d fini\n", i);
	}
	p_height = (N+BK_NUM-1) / BK_NUM;
	q_height = (M+BK_NUM-1)/BK_NUM;
	t_p_height = (p_height+TD_NUM-1)/TD_NUM;
	t_q_height = (q_height+TD_NUM-1)/TD_NUM;

	map<long, double>::iterator it; 
	for(it=EntryM.begin(); it!=EntryM.end(); it++){
		long hash_idx = it->first;
		double ra = it->second;
		int p = (int)(hash_idx/M);
		int q = (int)(hash_idx%M);
		int b_p = p/p_height;
		int b_q = q/q_height;
		int b_t_p = b_p/t_p_height;
		int b_t_q = b_q/t_q_height;
		Rblocks[b_t_p][b_t_q].push_back(RatingEntry(p,q,ra));
	}


	/*
	for (i = 0; i < BK_NUM; i++)
	{
		long row_sum = 0;
		printf("[%d]\t", i);
		for (j = 0; j < BK_NUM; j++)
		{
			printf("%ld\t", Rblocks[i][j].size());
			row_sum += Rblocks[i][j].size();
		}
		printf("{%ld}\n", row_sum );
	}
	**/

}
void initParas()
{
	partitionP(BK_NUM, N, PBlocks);
	partitionP(BK_NUM, M, QBlocks);
	int i, j;
	for (i = 0; i < BK_NUM; i++)
	{
		for (j = 0; j < PBlocks[i].ele_num; j++)
		{
			PBlocks[i].eles[j] = drand48() * 0.2;
		}
		for (j = 0; j < QBlocks[i].ele_num; j++)
		{
			QBlocks[i].eles[j] = drand48() * 0.2;
		}
	}
	for (i = 0; i < ITER_CAP; i++)
	{
		for (j = 0; j < TD_NUM; j++)
		{
			random_seq[i * TD_NUM + j] = j;
		}
	}
	for (i = 0; i < ITER_CAP; i++)
	{
		random_shuffle(random_seq + i * TD_NUM, random_seq + (i + 1)*TD_NUM );
	}
	printf("debug...\n");
	for(i = 0; i<10; i++){
		printf("%d\t", random_seq[i]);
	}

}
void allocCudaMem()
{
	HANDLE_ERROR(hipMalloc((void**)&dev_PBlocks, sizeof(Block)* BK_NUM));
	HANDLE_ERROR(hipMalloc((void**)&dev_QBlocks, sizeof(Block)* BK_NUM));
	int i = 0;
	for ( i = 0; i < BK_NUM; i++)
	{
		HANDLE_ERROR(hipMalloc((void**) &(dev_PData[i]), sizeof(double) * (PBlocks[i].ele_num)));
		HANDLE_ERROR(hipMalloc((void**) &(dev_QData[i]), sizeof(double) * (QBlocks[i].ele_num)));
	}

	HANDLE_ERROR(hipMalloc((void**)&dev_seq, sizeof(int) * (TD_NUM * ITER_CAP)) );
	HANDLE_ERROR(hipMalloc((void**)&dev_flag, sizeof(int) * (BK_NUM)) );

	int j =0;
	int idx = 0;
	for(i = 0; i<BT_NUM; i++){
		for(j = 0; j < BT_NUM; j++){
			idx = i*(BT_NUM)+j;
			entry_num[idx] = Rblocks[i][j].size();
			HANDLE_ERROR(hipMalloc((void**)&(dev_rate_entries[idx]), sizeof(RatingEntry) * (entry_num[idx])) );
		}
	}
	for(i = 0; i<BT_NUM; i++){
		HANDLE_ERROR(hipMalloc((void**)&(dev_p_cache[i]), sizeof(double) * (K)) );
		HANDLE_ERROR(hipMalloc((void**)&(dev_q_cache[i]), sizeof(double) * (K)) );
	}

	//should be dynamic
	HANDLE_ERROR(hipMemcpy( (dev_PBlocks), PBlocks, sizeof(Block) * (BK_NUM), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy( (dev_QBlocks), QBlocks, sizeof(Block) * (BK_NUM), hipMemcpyHostToDevice));
	for (i = 0; i < BK_NUM; i++)
	{
		HANDLE_ERROR(hipMemcpy( (dev_PData[i]), PBlocks[i].eles, sizeof(double) * (PBlocks[i].ele_num), hipMemcpyHostToDevice));
		HANDLE_ERROR(hipMemcpy( (dev_QData[i]), QBlocks[i].eles, sizeof(double) * (QBlocks[i].ele_num), hipMemcpyHostToDevice));
	}


	HANDLE_ERROR(hipMemcpy( (dev_seq), random_seq, sizeof(int) * (TD_NUM * ITER_CAP), hipMemcpyHostToDevice));

////
	HANDLE_ERROR(hipMemcpy( (dev_entry_num), entry_num, sizeof(int) * (RB_NUM), hipMemcpyHostToDevice));
////

	for(i = 0; i<BT_NUM; i++){
		for(j = 0; j < BT_NUM; j++){
			idx = i*(BT_NUM)+j;
			if(Rblocks[i][j].size()> 0){
					RatingEntry* cache_re = (RatingEntry*)malloc(sizeof(RatingEntry)*Rblocks[i][j].size());
					int k = 0;
					for(k =0; k < Rblocks[i][j].size(); k++){
						cache_re[k] = Rblocks[i][j][k];
					}
					HANDLE_ERROR(hipMemcpy( (dev_rate_entries[idx]), cache_re, sizeof(RatingEntry) * (Rblocks[i][j].size()), hipMemcpyHostToDevice));
					free(cache_re);
			}
			
		}
	}



}
void freeCudaMem()
{
	int i, j;
	hipFree(dev_PBlocks);
	hipFree(dev_QBlocks);
	for ( i = 0; i < BK_NUM; i++)
	{
		hipFree(dev_PData[i]);
		hipFree(dev_QData[i]);
	}
	hipFree(dev_seq);
	hipFree(dev_flag);

	for(i = 0; i<BT_NUM; i++){
		for(j = 0; j < BT_NUM; j++){
			hipFree(dev_rate_entries[i*BT_NUM+j]);
		}
	}
	for(i = 0; i<BT_NUM; i++){
		hipFree(dev_p_cache[i]);
		hipFree(dev_q_cache[i]);
	}
}
void partitionP(int portion_num, int line_num,  Block * block_arr)
{
	int i = 0;
	int height = (line_num+portion_num-1) / portion_num;
	//int last_height = N - (portion_num - 1) * height;

	for (i = 0; i < portion_num; i++)
	{
		block_arr[i].block_id = i;
		block_arr[i].data_age = 0;
		block_arr[i].height = height;
		int sta_idx = i * height;
		/*
		if ( i == portion_num - 1)
		{
			block_arr[i].height = last_height;
		}
		**/
		block_arr[i].sta_idx = sta_idx;
		block_arr[i].ele_num = block_arr[i].height * K;
		block_arr[i].eles = Malloc(double, block_arr[i].ele_num);
	}

}

int main(void)
{
	readTrainData();
	printf("readTrainData Fini\n");
	initParas();
	printf("initParas Fini\n");
	allocCudaMem();
	printf("allocCudaMem Fini\n");
	getchar();
	MFkernel <<< SM_NUM, 1>>>(dev_PBlocks, dev_QBlocks, dev_PData, dev_QData, dev_p_cache,dev_q_cache,dev_seq, dev_entry_num, dev_rate_entries, p_height, q_height, dev_flag, 0, yita, theta);
	printf("MFkernel Fini\n");
	hipDeviceSynchronize();
	printf("hipDeviceSynchronize Fini\n");
	freeCudaMem();
	printf("freeCudaMem Fini\n");
	//cudaDeviceReset();
	return 0;
}
